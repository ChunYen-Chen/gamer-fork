#include "CUAPI.h"
#include "CUFLU.h"

#ifdef GPU



// *******************************************
// ** CUDA stream objects are declared here **
hipStream_t *Stream;
// *******************************************


extern real (*d_Flu_Array_F_In )[FLU_NIN ][ CUBE(FLU_NXT) ];
extern real (*d_Flu_Array_F_Out)[FLU_NOUT][ CUBE(PS2) ];
extern real (*d_Flux_Array)[9][NFLUX_TOTAL][ SQR(PS2) ];
#ifdef UNSPLIT_GRAVITY
extern real (*d_Pot_Array_USG_F)[ CUBE(USG_NXT_F) ];
#endif
extern double (*d_Corner_Array_F)[3];
#ifdef DUAL_ENERGY
extern char (*d_DE_Array_F_Out)[ CUBE(PS2) ];
#endif

// global memory arrays in different models
#if ( MODEL == HYDRO )
#if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
extern real (*d_PriVar)      [NCOMP_TOTAL][ CUBE(FLU_NXT)     ];
extern real (*d_Slope_PPM)[3][NCOMP_TOTAL][ CUBE(N_SLOPE_PPM) ];
extern real (*d_FC_Var)   [6][NCOMP_TOTAL][ CUBE(N_FC_VAR)    ];
extern real (*d_FC_Flux)  [3][NCOMP_TOTAL][ CUBE(N_FC_FLUX)   ];
#endif // #if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )

#elif ( MODEL == MHD )
#warning : WAIT MHD !!!

#elif ( MODEL != ELBDM )
#warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#endif // MODEL




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemAllocate_Fluid
// Description :  Allocate GPU and CPU memory for the fluid solver
//
// Parameter   :  Flu_NPG     : Number of patch groups evaluated simultaneously by GPU for the fluid solver
//                GPU_NStream : Number of CUDA stream objects
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemAllocate_Fluid( const int Flu_NPG, const int GPU_NStream )
{

// determine whether or not to allocate the corner array
   bool AllocateCorner = false;

#  ifdef UNSPLIT_GRAVITY
   if ( OPT__GRAVITY_TYPE == GRAVITY_EXTERNAL  ||  OPT__GRAVITY_TYPE == GRAVITY_BOTH )
      AllocateCorner = true;
#  endif

#  if ( COORDINATE != CARTESIAN )
      AllocateCorner = true;
#  endif


// size of the global memory arrays in all models
   const long Flu_MemSize_F_In  = sizeof(real  )*Flu_NPG*FLU_NIN *FLU_NXT*FLU_NXT*FLU_NXT;
   const long Flu_MemSize_F_Out = sizeof(real  )*Flu_NPG*FLU_NOUT*PS2*PS2*PS2;
   const long Flux_MemSize      = sizeof(real  )*Flu_NPG*9*NFLUX_TOTAL*PS2*PS2;
#  ifdef UNSPLIT_GRAVITY
   const long Pot_MemSize_USG_F = sizeof(real  )*Flu_NPG*USG_NXT_F*USG_NXT_F*USG_NXT_F;
#  endif
   const long Corner_MemSize_F  = ( AllocateCorner ) ? sizeof(double)*Flu_NPG*3 : 0;
#  ifdef DUAL_ENERGY
   const long DE_MemSize_F_Out  = sizeof(char  )*Flu_NPG*PS2*PS2*PS2;
#  endif

// the size of the global memory arrays in different models
#  if   ( MODEL == HYDRO )
#  if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
   const long PriVar_MemSize    = Flu_MemSize_F_In;
   const long FC_Var_MemSize    = sizeof(real)*Flu_NPG*6*NCOMP_TOTAL*CUBE(N_FC_VAR);
   const long FC_Flux_MemSize   = sizeof(real)*Flu_NPG*3*NCOMP_TOTAL*CUBE(N_FC_FLUX);

#  if ( LR_SCHEME == PPM )
   const long Slope_PPM_MemSize = sizeof(real)*Flu_NPG*3*NCOMP_TOTAL*CUBE(N_SLOPE_PPM);
#  endif

#  endif // #if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )

#  elif ( MODEL == MHD )
#  warning : WAIT MHD !!!

#  elif ( MODEL != ELBDM )
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif // MODEL


// output the total memory requirement
   long TotalSize = Flu_MemSize_F_In + Flu_MemSize_F_Out;

   if ( amr->WithFlux )
   TotalSize += Flux_MemSize;

#  ifdef UNSPLIT_GRAVITY
   TotalSize += Pot_MemSize_USG_F;
#  endif

   if ( AllocateCorner )
   TotalSize += Corner_MemSize_F;

#  ifdef DUAL_ENERGY
   TotalSize += DE_MemSize_F_Out;
#  endif

#  if   ( MODEL == HYDRO )

#  if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
   TotalSize += PriVar_MemSize + FC_Var_MemSize + FC_Flux_MemSize;

#  if ( LR_SCHEME == PPM )
   TotalSize += Slope_PPM_MemSize;
#  endif // PPM
#  endif // MHM/MHM_RP/CTU

#  elif ( MODEL == MHD )
#  warning : WAIT MHD !!!

#  elif ( MODEL != ELBDM )
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif // MODEL

   if ( MPI_Rank == 0 )
      Aux_Message( stdout, "NOTE : total memory requirement in GPU fluid solver = %ld MB\n", TotalSize/(1<<20) );


// allocate the device memory (in all models)
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Flu_Array_F_In,        Flu_MemSize_F_In        )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Flu_Array_F_Out,       Flu_MemSize_F_Out       )  );

   if ( amr->WithFlux )
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Flux_Array,            Flux_MemSize            )  );

#  ifdef UNSPLIT_GRAVITY
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Pot_Array_USG_F,       Pot_MemSize_USG_F       )  );
#  endif

   if ( AllocateCorner )
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Corner_Array_F,        Corner_MemSize_F        )  );


// allocate the device memory (in different models)
#  if   ( MODEL == HYDRO )
#  ifdef DUAL_ENERGY
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_DE_Array_F_Out,        DE_MemSize_F_Out        )  );
#  endif

#  if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Var,                FC_Var_MemSize          )  );

   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Flux,               FC_Flux_MemSize         )  );

   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_PriVar,                PriVar_MemSize          )  );

#  if ( LR_SCHEME == PPM )
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Slope_PPM,             Slope_PPM_MemSize       )  );
#  endif
#  endif // #if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )

#  elif ( MODEL == MHD )
#  warning : WAIT MHD !!!

#  elif ( MODEL != ELBDM )
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif // MODEL


// allocate the host memory by CUDA
   for (int t=0; t<2; t++)
   {
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Flu_Array_F_In [t], Flu_MemSize_F_In        )  );
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Flu_Array_F_Out[t], Flu_MemSize_F_Out       )  );

      if ( amr->WithFlux )
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Flux_Array     [t], Flux_MemSize            )  );

#     ifdef UNSPLIT_GRAVITY
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Pot_Array_USG_F[t], Pot_MemSize_USG_F       )  );
#     endif

      if ( AllocateCorner )
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Corner_Array_F [t], Corner_MemSize_F        )  );

#     ifdef DUAL_ENERGY
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_DE_Array_F_Out [t], DE_MemSize_F_Out        )  );
#     endif
   } // for (int t=0; t<2; t++)


// create streams
   Stream = new hipStream_t [GPU_NStream];
   for (int s=0; s<GPU_NStream; s++)      CUDA_CHECK_ERROR(  hipStreamCreate( &Stream[s] )  );

} // FUNCTION : CUAPI_MemAllocate_Fluid



#endif // #ifdef GPU
