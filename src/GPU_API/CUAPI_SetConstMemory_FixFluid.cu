#include "CUAPI.h"
#include "CUDA_ConstMemory.h"

#if ( defined GPU )

//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_SetConstMemory_FixFluid
// Description :  Set the fix fluid constant memory variables on GPU
//
// Note        :  1. Adopt the suggested approach for CUDA version >= 5.0
//                2. Invoked by Init_Load_FixFluid()
//
// Parameter   :  None
//
// Return      :  c_FixSwitch[]
//                FixFlu.FixSwitchPtr
//---------------------------------------------------------------------------------------------------
void CUAPI_SetConstMemory_FixFluid()
{

// copy data to constant memory
   CUDA_CHECK_ERROR(  hipMemcpyToSymbol(HIP_SYMBOL( c_FixSwitch), FixFlu.FixSwitch, (NCOMP_TOTAL+NCOMP_MAG)*sizeof(int   ) )  );

// obtain the constant-memory pointers
   CUDA_CHECK_ERROR(  hipGetSymbolAddress( (void **)&FixFlu.FixSwitchPtr, c_FixSwitch )   );

} // FUNCTION : CUAPI_SetConstMemory_FixFluid



#endif // #if ( defined GPU )
