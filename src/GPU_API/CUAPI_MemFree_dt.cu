#include "CUAPI.h"

#ifdef GPU



extern real    *d_dt_Array_T;
extern real   (*d_Flu_Array_T)[NCOMP_FLUID][ CUBE(PS1) ];
extern double (*d_Corner_Array_T)[3];
#ifdef GRAVITY
extern real   (*d_Pot_Array_T)[ CUBE(GRA_NXT) ];
#endif




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemFree_dt
// Description :  Free the GPU and CPU memory previously allocated by CUAPI_MemAllocate_dt()
//
// Parameter   :  None
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemFree_dt()
{

// free the device memory
   if ( d_dt_Array_T     != NULL ) {   CUDA_CHECK_ERROR(  hipFree( d_dt_Array_T     )  );   d_dt_Array_T     = NULL; }
   if ( d_Flu_Array_T    != NULL ) {   CUDA_CHECK_ERROR(  hipFree( d_Flu_Array_T    )  );   d_Flu_Array_T    = NULL; }
   if ( d_Corner_Array_T != NULL ) {   CUDA_CHECK_ERROR(  hipFree( d_Corner_Array_T )  );   d_Corner_Array_T = NULL; }
#  ifdef GRAVITY
   if ( d_Pot_Array_T    != NULL ) {   CUDA_CHECK_ERROR(  hipFree( d_Pot_Array_T    )  );   d_Pot_Array_T    = NULL; }
#  endif


// free the host memory allocated by CUDA
   for (int t=0; t<2; t++)
   {
      if ( h_dt_Array_T    [t] != NULL ) {   CUDA_CHECK_ERROR(  hipHostFree( h_dt_Array_T    [t] )  );  h_dt_Array_T    [t] = NULL; }
      if ( h_Flu_Array_T   [t] != NULL ) {   CUDA_CHECK_ERROR(  hipHostFree( h_Flu_Array_T   [t] )  );  h_Flu_Array_T   [t] = NULL; }
      if ( h_Corner_Array_T[t] != NULL ) {   CUDA_CHECK_ERROR(  hipHostFree( h_Corner_Array_T[t] )  );  h_Corner_Array_T[t] = NULL; }
#     ifdef GRAVITY
      if ( h_Pot_Array_T   [t] != NULL ) {   CUDA_CHECK_ERROR(  hipHostFree( h_Pot_Array_T   [t] )  );  h_Pot_Array_T   [t] = NULL; }
#     endif
   }

} // FUNCTION : CUAPI_MemFree_dt


#endif // #ifdef GPU
